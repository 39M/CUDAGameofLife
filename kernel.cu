#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <assert.h>
#include <algorithm>

#include "pre_define.h"

#define CUDA_CALL(__func__) {const hipError_t __cuda_err__ = (__func__); if (__cuda_err__ != hipSuccess) {printf("\nCuda Error: %s (err_num=%d)\n", hipGetErrorString(__cuda_err__), __cuda_err__); hipDeviceReset(); assert(0);}}

/********** Version 1 **********/
// Use symbol(__device__ variables as below) to store cells status


__device__ char gpu_cells[CELL_X + 2][CELL_Y + 2];
__device__ char gpu_cells_next[CELL_X + 2][CELL_Y + 2];


// Kernel
__global__ void simpleUpdateKernel()
{
	int i = blockIdx.x + 1;
	int j = threadIdx.x + 1;
	int cellsCount = gpu_cells[i - 1][j - 1] + gpu_cells[i - 1][j] + gpu_cells[i - 1][j + 1] +
		gpu_cells[i][j - 1] + gpu_cells[i][j + 1] +
		gpu_cells[i + 1][j - 1] + gpu_cells[i + 1][j] + gpu_cells[i + 1][j + 1];

	if (cellsCount == 3)
		gpu_cells_next[i][j] = 1;
	else if (cellsCount == 2)
		gpu_cells_next[i][j] = gpu_cells[i][j];
	else
		gpu_cells_next[i][j] = 0;
}


// A not efficient kernel
/*__global__ void updateKernelPlus()
{
	int i = blockIdx.x / (CELL_Y / BLOCK_DIM) + 1;
	int j = threadIdx.x + BLOCK_DIM * (blockIdx.x % (CELL_Y / BLOCK_DIM)) + 1;
	int cellsCount = gpu_cells[i - 1][j - 1] + gpu_cells[i - 1][j] + gpu_cells[i - 1][j + 1] +
		gpu_cells[i][j - 1] + gpu_cells[i][j + 1] +
		gpu_cells[i + 1][j - 1] + gpu_cells[i + 1][j] + gpu_cells[i + 1][j + 1];

	if (cellsCount == 3)
		gpu_cells_next[i][j] = 1;
	else if (cellsCount == 2)
		gpu_cells_next[i][j] = gpu_cells[i][j];
	else
		gpu_cells_next[i][j] = 0;
}*/


// Copy CPU cells status to GPU and call kernel, then copy data back
extern "C" int CUDAUpdate(char cells[CELL_X + 2][CELL_Y + 2], int iterateTime)
{
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_cells), cells, (CELL_X + 2) * (CELL_Y + 2)));
	for (int iterator = 0; iterator < iterateTime; iterator++)
	{
		simpleUpdateKernel << <CELL_X, CELL_Y >> >();
		//updateKernelPlus << < CELL_X * (CELL_Y / BLOCK_DIM), BLOCK_DIM >> >();

		// Why not copy data form GPU to GPU directly?
		// But there's not a function "cudaMemcpyFromSymbolToSymbol"
		// This makes it less efficient
		CUDA_CALL(hipMemcpyFromSymbol(cells, HIP_SYMBOL(gpu_cells_next), (CELL_X + 2) * (CELL_Y + 2)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_cells), cells, (CELL_X + 2) * (CELL_Y + 2)));

		// This function cannot use for symbol
		//CUDA_CALL(hipMemcpy(gpu_cells, gpu_cells_next, (CELL_X + 2) * (CELL_Y + 2), hipMemcpyDeviceToDevice));
	}
	CUDA_CALL(hipMemcpyFromSymbol(cells, HIP_SYMBOL(gpu_cells), (CELL_X + 2) * (CELL_Y + 2)));
	return 0;
}
/********** Version 1 End **********/


/********** Version 2 **********/
// Alloc GPU memory to store cells status


// Kernel
__global__ void anotherSimpleUpdateKernel(char *gpu_cells, char *gpu_cells_next)
{
	int i = blockIdx.x + 1;
	int j = threadIdx.x + 1;
	int cellsCount = gpu_cells[(i - 1) * (CELL_Y + 2) + j - 1] + gpu_cells[(i - 1) * (CELL_Y + 2) + j] + gpu_cells[(i - 1) * (CELL_Y + 2) + j + 1] +
		gpu_cells[i * (CELL_Y + 2) + j - 1] + gpu_cells[i * (CELL_Y + 2) + j + 1] +
		gpu_cells[(i + 1) * (CELL_Y + 2) + j - 1] + gpu_cells[(i + 1) * (CELL_Y + 2) + j] + gpu_cells[(i + 1) * (CELL_Y + 2) + j + 1];

	if (cellsCount == 3)
		gpu_cells_next[i * (CELL_Y + 2) + j] = 1;
	else if (cellsCount == 2)
		gpu_cells_next[i * (CELL_Y + 2) + j] = gpu_cells[i * (CELL_Y + 2) + j];
	else
		gpu_cells_next[i * (CELL_Y + 2) + j] = 0;
}


// Copy CPU cells status to GPU and call kernel, then copy data back
extern "C" int anotherCUDAUpdate(char cells[CELL_X + 2][CELL_Y + 2], int iterateTime)
{
	char *gpu_cells_pointer;
	char *gpu_cells_next_pointer;

	CUDA_CALL(hipMalloc((void**)&gpu_cells_pointer, (CELL_X + 2) * (CELL_Y + 2)));
	CUDA_CALL(hipMalloc((void**)&gpu_cells_next_pointer, (CELL_X + 2) * (CELL_Y + 2)));

	CUDA_CALL(hipMemcpy(gpu_cells_pointer, cells, (CELL_X + 2) * (CELL_Y + 2), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemset(gpu_cells_next_pointer, 0, (CELL_X + 2) * (CELL_Y + 2)));

	for (int iterator = 0; iterator < iterateTime; iterator++)
	{
		anotherSimpleUpdateKernel << <CELL_X, CELL_Y >> >(gpu_cells_pointer, gpu_cells_next_pointer);
		//CUDA_CALL(hipMemcpy(gpu_cells_pointer, gpu_cells_next_pointer, (CELL_X + 2) * (CELL_Y + 2), hipMemcpyDeviceToDevice));
		std::swap(gpu_cells_pointer, gpu_cells_next_pointer);
	}
	CUDA_CALL(hipMemcpy(cells, gpu_cells_pointer, (CELL_X + 2) * (CELL_Y + 2), hipMemcpyDeviceToHost));

	CUDA_CALL(hipFree(gpu_cells_pointer));
	CUDA_CALL(hipFree(gpu_cells_next_pointer));
	return 0;
}
/********** Version 2 End **********/
